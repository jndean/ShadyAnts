#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>


#define PI 3.141592653589793


// Agents
#define NUM_AGENTS 1000
#define SPEED 1.5f

// Trails
#define DIFFUSE_RAD 1
#define DECAY 0.7f
#define FLUX 0.2f


struct Agent {
    float x, y;
    float angle;
    uint32_t state;
};


Agent* d_agents;
float* d_trails_back;
float* d_trails_front;


// ------------------------ HELPERS ------------------------ //

__device__ float clamp(float x, float a, float b)
{
    return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b)
{
    return max(a, min(b, x));
}

__device__ int rgbToInt(float r, float g, float b)
{
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b)<<16) | (int(g)<<8) | int(r);
}

inline __device__
uint32_t step_state(uint32_t state)
{
    state ^= 2747636419u;
    state *= 2654435769u;
    state ^= state >> 16;
    state *= 2654435769u;
    state ^= state >> 16;
    state *= 2654435769u;
    return state;
}



// ------------------------------ SETUP ------------------------------ //



extern "C"
__host__ void CUDAinit(unsigned int** texture, int imgw, int imgh, int ubyte_size)
{
    unsigned int texture_size = imgw * imgh * 4 * ubyte_size;
    checkCudaErrors(hipMalloc((void **)texture, texture_size));
    
    Agent h_agents[NUM_AGENTS];
    srand(6);
    for (int i = 0; i < NUM_AGENTS; ++i) {
	h_agents[i].x = rand() % imgw;
	h_agents[i].y = rand() % imgh;
	h_agents[i].state = rand();
	h_agents[i].angle = ((float)h_agents[i].state) / RAND_MAX * PI * 2;
    }
    checkCudaErrors(hipMalloc(&d_agents, NUM_AGENTS * sizeof(Agent)));
    checkCudaErrors(hipMalloc(&d_trails_back, imgw * imgh * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_trails_front, imgw * imgh * sizeof(float)));
    checkCudaErrors(hipMemcpy(d_agents, h_agents, NUM_AGENTS * sizeof(Agent), hipMemcpyHostToDevice));
    hipDeviceSynchronize();
}


// ------------------------------ KERNELS ------------------------------ //


__global__ void
moveAgents(Agent* agents, float *trails, float imgw, float imgh)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= NUM_AGENTS) return;

    Agent agent = agents[i];

    agent.x += sin(agent.angle) * SPEED;
    agent.y += cos(agent.angle) * SPEED;
    if (agent.x < 0 || agent.x >= imgw || agent.y < 0 || agent.y >= imgh) {
	agent.state = step_state(agent.state);
	agent.angle = agent.state / 1000.0f;
	agent.x = clamp(agent.x, 0.0f, imgw-1);
	agent.y = clamp(agent.y, 0.0f, imgh-1);
    }
    
    trails[((int) agent.y) * (int)imgw + ((int) agent.x)] = 255.0f;
    agents[i] = agent;
}




__global__ void
diffuse(float* trails_front, float* trails_back, unsigned int *g_odata, int imgw, int imgh)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= imgw || y >= imgh) return;

    float val = trails_front[y * imgw + x];
    g_odata[y*imgw+x] = rgbToInt(val, val, val);
    
    float sum = 0;
    for (int dy = -DIFFUSE_RAD; dy < DIFFUSE_RAD; ++dy) {
	for (int dx = -DIFFUSE_RAD; dx < DIFFUSE_RAD; ++dx) {
	    int sample_y = clamp(y+dy, 0, imgh);
	    int sample_x = clamp(x+dx, 0, imgw);
	    sum += trails_front[sample_y * imgw + sample_x];
	}
    }
    sum /= 2; //DIFFUSE_AREA;
    
    trails_back[y*imgw+x] = FLUX * sum + (1-FLUX) * val * DECAY;
}



extern "C" void
step_simulation(unsigned int *odata, int imgw, int imgh)
{

    int block_size = 32;
    int num_blocks = (NUM_AGENTS + block_size - 1) / block_size;
    moveAgents<<<num_blocks, block_size>>>(d_agents, d_trails_front, imgw, imgh);

    
    dim3 block_dim(block_size, block_size);
    dim3 grid_dim((imgw + block_size - 1) / block_size, (imgh + block_size - 1) / block_size);
    diffuse<<<grid_dim, block_dim>>>(d_trails_front, d_trails_back, odata, imgw, imgh);

    std::swap(d_trails_front, d_trails_back);
    
}